
#include "Core/EDXNet.h"

using namespace EDX;
using namespace DeepLearning;
using namespace Algorithm;

void TestFullyConnectedCUDA()
{
	int numInputs = 2;
	int numHidden = 3;
	TensorShape inputShape = { 4, 5, 6 };

	int inputSize = numInputs * Algorithm::Accumulate(inputShape, 1, Algorithm::Multiply<>());
	int weightSize = numHidden * Algorithm::Accumulate(inputShape, 1, Algorithm::Multiply<>());

	Tensorf x = Tensorf::LinSpace(-0.1f, 0.5f, inputSize).Reshape(numInputs, inputSize / numInputs);
	Tensorf w = Tensorf::LinSpace(-0.2f, 0.3f, weightSize).Reshape(inputSize / numInputs, numHidden);
	Tensorf b = Tensorf::LinSpace(-0.3f, 0.1f, numHidden);

	Tensorf r = Tensorf::Dot(x, w);

	Symbol* data = NeuralNet::Create<Variable>(x);
	Symbol* weights = NeuralNet::Create<Variable>(w);
	Symbol* biases = NeuralNet::Create<Variable>(b);

	Symbol* fullyConnected = NeuralNet::Create<FullyConnected>(data, weights, biases, numHidden);

	NeuralNet net(fullyConnected, true);

	Array<Symbol*> symbolsToEvaluate = net.GetGradientSymbols({ data, weights, biases });
	symbolsToEvaluate.Add(fullyConnected); // Add loss layer

	net.Execute(symbolsToEvaluate);

	Tensorf result = fullyConnected->GetOutput();

	Tensorf correctResult = Tensorf({ { 1.49834967f, 1.70660132f, 1.91485297f },
										{ 3.25553199f, 3.5141327f, 3.77273342f } });

	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(result - correctResult))) << "\n";

	Tensorf& dataGrad = symbolsToEvaluate[0]->GetOutput(data->GetGradientIndex());
	Tensorf& weightsGrad = symbolsToEvaluate[1]->GetOutput(weights->GetGradientIndex());
	Tensorf& biasesGrad = symbolsToEvaluate[2]->GetOutput(biases->GetGradientIndex());

	Tensorf upperGrads = TensorExpr::Ones(result.Shape());
	Tensorf dataNumericalGrad = NumericalGradientEval([&]() -> Tensorf
	{
		net.Execute({ fullyConnected });
		return fullyConnected->GetOutput();
	},
		data, upperGrads);
	Tensorf numericalWeightsGrad = NumericalGradientEval([&]() -> Tensorf
	{
		net.Execute({ fullyConnected });
		return fullyConnected->GetOutput();
	},
		weights, upperGrads);
	Tensorf numericalBiasesGrad = NumericalGradientEval([&]() -> Tensorf
	{
		net.Execute({ fullyConnected });
		return fullyConnected->GetOutput();
	},
		biases, upperGrads);

	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(dataGrad - dataNumericalGrad))) << "\n";
	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(weightsGrad - numericalWeightsGrad))) << "\n";
	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(biasesGrad - numericalBiasesGrad))) << "\n";

	NeuralNet::Release();
}

bool TestConvolutionCUDA()
{
	TensorShape x_shape = { 2, 3, 4, 4 };
	TensorShape w_shape = { 3, 3, 4, 4 };

	int xSize = Algorithm::Accumulate(x_shape, 1, Algorithm::Multiply<>());
	int wSize = Algorithm::Accumulate(w_shape, 1, Algorithm::Multiply<>());

	Tensorf x = Tensorf::LinSpace(-0.1f, 0.5f, xSize).Reshape(x_shape);
	Tensorf w = Tensorf::LinSpace(-0.2f, 0.3f, wSize).Reshape(w_shape);
	Tensorf b = Tensorf::LinSpace(-0.1f, 0.2f, 3);

	Symbol* data = NeuralNet::Create<Variable>(x);
	Symbol* weights = NeuralNet::Create<Variable>(w);
	Symbol* biases = NeuralNet::Create<Variable>(b);

	Symbol* convolution = NeuralNet::Create<Convolution>(data, weights, biases, TensorShape({ 4, 4 }), 3, TensorShape({ 2, 2 }), TensorShape({ 1, 1 }));

	NeuralNet net(convolution, true);

	Array<Symbol*> symbolsToEvaluate = net.GetGradientSymbols({ data, weights, biases });
	symbolsToEvaluate.Add(convolution); // Add loss layer

	net.Execute(symbolsToEvaluate);

	Tensorf result = convolution->GetOutput();
	Tensorf correctResult = Tensorf({ { { { -0.08759809f, -0.10987781f },
			{-0.18387192f, -0.2109216f}},
			{{0.21027089f, 0.21661097f},
			{0.22847626f, 0.23004637f}},
			{{0.50813986f, 0.54309974f},
			{0.64082444f, 0.67101435f}}},
			{{{-0.98053589f, -1.03143541f},
			{-1.19128892f, -1.24695841f}},
			{{0.69108355f, 0.66880383f},
			{0.59480972f, 0.56776003f}},
			{{2.36270298f, 2.36904306f},
			{2.38090835f, 2.38247847f}}} });

	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(result - correctResult))) << "\n";

	Tensorf& dataGrad = symbolsToEvaluate[0]->GetOutput(data->GetGradientIndex());
	Tensorf& weightsGrad = symbolsToEvaluate[1]->GetOutput(weights->GetGradientIndex());
	Tensorf& biasesGrad = symbolsToEvaluate[2]->GetOutput(biases->GetGradientIndex());

	Tensorf upperGrads = TensorExpr::Ones(result.Shape());
	Tensorf dataNumericalGrad = NumericalGradientEval([&]() -> Tensorf
	{
		net.Execute({ convolution });
		return convolution->GetOutput();
	},
		data, upperGrads);
	Tensorf numericalWeightsGrad = NumericalGradientEval([&]() -> Tensorf
	{
		net.Execute({ convolution });
		return convolution->GetOutput();
	},
		weights, upperGrads);
	Tensorf numericalBiasesGrad = NumericalGradientEval([&]() -> Tensorf
	{
		net.Execute({ convolution });
		return convolution->GetOutput();
	},
		biases, upperGrads);

	std::cout << Tensorf::Sum(TensorExpr::Abs(dataGrad - dataNumericalGrad)) << "\n";
	std::cout << Tensorf::Sum(TensorExpr::Abs(weightsGrad - numericalWeightsGrad)) << "\n";
	std::cout << Tensorf::Sum(TensorExpr::Abs(biasesGrad - numericalBiasesGrad)) << "\n";

	NeuralNet::Release();

	return true;
}

bool TestReluCUDA()
{
	Tensorf input = Tensorf::LinSpace(-0.5, 0.5, 12).Reshape(3, 4);

	Symbol* x = NeuralNet::Create<Variable>(input);
	Symbol* relu = NeuralNet::Create<Relu>(x);

	NeuralNet net(relu, true);

	Array<Symbol*> symbolsToEvaluate = net.GetGradientSymbols({ x });
	symbolsToEvaluate.Add(relu); // Add loss layer

	net.Execute(symbolsToEvaluate);

	Tensorf result = relu->GetOutput();
	Tensorf correctResult = Tensorf({ { 0.0f, 0.0f, 0.0f, 0.0f, },
									{ 0.0f, 0.0f, 0.04545455f, 0.13636364f, },
									{ 0.22727273f, 0.31818182f, 0.40909091f, 0.5f, } });

	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(result - correctResult))) << "\n";

	Tensorf& dx = symbolsToEvaluate[0]->GetOutput(0);

	Tensorf upperGrads = TensorExpr::Ones(result.Shape());
	Tensorf dxNumerical = NumericalGradientEval([&]() -> Tensorf
		{
			net.Execute({ relu });
			return relu->GetOutput();
		},
		x, upperGrads);

	std::cout << Tensorf::Sum(Tensorf(TensorExpr::Abs(dx - dxNumerical))) << "\n";

	NeuralNet::Release();

	return true;
}


void TestPoolingCUDA()
{
	TensorShape xShape = { 2,3,4,4 };
	Tensorf input = Tensorf::LinSpace(-0.3f, 0.4f, Algorithm::Accumulate(xShape, 1, Algorithm::Multiply<>())).Reshape(xShape);

	Symbol* x = NeuralNet::Create<Variable>(input);
	Symbol* pooling = NeuralNet::Create<MaxPooling>(x, TensorShape({ 2,2 }), TensorShape({ 2,2 }), TensorShape({ 0,0 }));

	NeuralNet net(pooling, true);

	Array<Symbol*> symbolsToEvaluate = net.GetGradientSymbols({ x });
	symbolsToEvaluate.Add(pooling); // Add loss layer

	net.Execute(symbolsToEvaluate);

	Tensorf& result = pooling->GetOutput();
	Tensorf correctResult = Tensorf({ { { { -0.26315789f, -0.24842105f },
											{ -0.20421053f, -0.18947368f } },
											{ { -0.14526316f, -0.13052632f },
											{ -0.08631579f, -0.07157895f } },
											{ { -0.02736842f, -0.01263158f },
											{ 0.03157895f, 0.04631579f } } },
											{ { { 0.09052632f, 0.10526316f },
											{ 0.14947368f, 0.16421053f } },
											{ { 0.20842105f, 0.22315789f },
											{ 0.26736842f, 0.28210526f } },
											{ { 0.32631579f, 0.34105263f },
											{ 0.38526316f, 0.4f } } } });

	std::cout << Tensorf::Sum(TensorExpr::Abs(result - correctResult)) << "\n";

	Tensorf& dx = symbolsToEvaluate[0]->GetOutput(0);

	Tensorf upperGrads = TensorExpr::Ones(result.Shape());
	Tensorf dxNumerical = NumericalGradientEval([&]() -> Tensorf
		{
			net.Execute({ pooling });
			return pooling->GetOutput();
		},
		x, upperGrads);

	std::cout << Tensorf::Sum(TensorExpr::Abs(dx - dxNumerical)) << "\n";

	NeuralNet::Release();
}


void TestSoftmaxCUDA()
{
	const int numClasses = 10;
	const int numInputs = 50;
	Tensorf dataX = Scalar(0.001f) * Tensorf::RandomNormalDistribution(1.0, numInputs, numClasses);
	Tensorf dataY = Tensorf::RandomInt(numClasses, numInputs);

	Symbol* x = NeuralNet::Create<Variable>(dataX);
	Symbol* y = NeuralNet::Create<Constant>(dataY);
	Symbol* softmax = NeuralNet::Create<Softmax>(x, y);

	NeuralNet net(softmax, true);

	Array<Symbol*> symbolsToEvaluate = net.GetGradientSymbols({ x });
	symbolsToEvaluate.Add(softmax); // Add loss layer

	net.Execute(symbolsToEvaluate);

	Tensorf result = softmax->GetOutput();
	Tensorf correctResult = 2.3f;

	std::cout << Tensorf::Sum(TensorExpr::Abs(result - correctResult)) << "\n";

	Tensorf& dx = symbolsToEvaluate[0]->GetOutput(0);

	Tensorf upperGrads = TensorExpr::Ones(result.Shape());
	Tensorf dxNumerical = NumericalGradientEval([&]() -> Tensorf
		{
			net.Execute({ softmax });
			return softmax->GetOutput();
		},
		x, upperGrads);

	std::cout << Tensorf::Sum(TensorExpr::Abs(dx - dxNumerical)) << "\n";

	NeuralNet::Release();
}

void TestCUDA()
{
	{
		Tensorf A = { 1,2,3,4,5,8,3,1,4 };
		Tensorf B = NestedInitializerList<float, 2>({ {1},{2},{3},{4},{5} });

		Tensorf C = A + B + A + A * B;

		float pHostC[45] = { 0 };
		hipMemcpy((void*)pHostC, (void*)C.Data(), C.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = { 9,2,3,4,5 };
		Tensorf B = { 9,2,3,4,5 };

		A += TensorExpr::Exp(B);

		float pHostA[5] = { 0 };
		hipMemcpy((void*)pHostA, (void*)A.Data(), A.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = { { 9,2,3,4,5 } };
		Tensorf B = NestedInitializerList<float, 2>({ {1},{2},{3},{4},{5} });

		Tensorf C = TensorExpr::Dot((A + B) * TensorExpr::Sum(Tensorf::LinSpace(0, 128, 128)), B) * B;

		float pHostA[25] = { 0 };
		hipMemcpy((void*)pHostA, (void*)C.Data(), C.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = Tensorf::LinSpace(0, 40960, 40960);
		Tensorf sum = TensorExpr::StandardDeviation(A);

		float pHost[1] = { 0 };
		hipMemcpy((void*)pHost, (void*)sum.Data(), sum.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = Tensorf::ArrayRange(0, 1000);
		A.Reshape(5, 10, 4, 5);

		Tensorf sum = TensorExpr::Sum(A, { 0, 2 }, true);

		float pHost[50] = { 0 };
		hipMemcpy((void*)pHost, (void*)sum.Data(), sum.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = { {9,2,3,4,5} };
		Tensorf B = NestedInitializerList<float, 2>({ {9},{2},{3},{4},{5} });

		Tensorf C = Tensorf::Dot(B, A);

		float pHost[25] = { 0 };
		hipMemcpy((void*)pHost, (void*)C.Data(), C.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	TestFullyConnectedCUDA();
	TestConvolutionCUDA();
	TestReluCUDA();
	TestPoolingCUDA();
	TestSoftmaxCUDA();
}