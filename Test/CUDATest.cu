
#include "Core/Tensor.h"

using namespace EDX;
using namespace DeepLearning;
using namespace Algorithm;

void TestCUDA()
{
	{
		Tensorf A = { 1,2,3,4,5,8,3,1,4 };
		Tensorf B = { 1,2,3,4,5 };

		B.Reshape(5, 1);

		Tensorf C = A + B + A + A * B;

		float pHostC[45] = { 0 };
		hipMemcpy((void*)pHostC, (void*)C.Data(), C.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = { 9,2,3,4,5 };
		Tensorf B = { 9,2,3,4,5 };

		A *= Tensorf::Exp(B);

		float pHostA[5] = { 0 };
		hipMemcpy((void*)pHostA, (void*)A.Data(), A.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}

	{
		Tensorf A = Tensorf::LinSpace(0, 40960, 40960);
		Tensorf sum = Tensorf::StandardDeviation(A);

		float pHost[1] = { 0 };
		hipMemcpy((void*)pHost, (void*)sum.Data(), sum.LinearSize() * sizeof(float), hipMemcpyDeviceToHost);
	}
}